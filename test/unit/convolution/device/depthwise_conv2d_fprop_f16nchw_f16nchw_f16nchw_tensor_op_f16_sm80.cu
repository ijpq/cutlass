
#include <hip/hip_runtime.h>
/***************************************************************************************************
 * Copyright (c) 2017-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 *modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice,
 *this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *notice, this list of conditions and the following disclaimer in the
 *documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its
 *contributors may be used to endorse or promote products derived from this
 *software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 *AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 *IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 *ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 *DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 *(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 *ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TOR
 *(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 *SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/**
 * \file
 * test/unit/convolution/device/depthwise_conv2d_fprop_f16nchw_f16nchw_f16nchw_tensor_op_f16_f32_sm70.cu
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or
 * implied.
 */
/*! \file
   \brief Tests for device-wide GEMM interface
*/

#if defined(__CUDACC__) && (__CUDACC_VER_MAJOR__ >= 11)
#include "cutlass/convolution/device/convolution.h"

#include "../../common/cutlass_unit_test.h"

#include "cutlass/util/reference/host/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/tensor_view_io.h"

#include "testbed.h"

#define RUN_DEPTHWISE_CONVOLUTION(stage)                                       \
    do {                                                                       \
        using ElementOutput = cutlass::half_t;                                 \
        using ElementAccumulator = cutlass::half_t;                            \
        using ElementBias = cutlass::half_t;                                   \
        using ElementCompute = cutlass::half_t;                                \
        using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;          \
        using Convolution = cutlass::conv::device::Convolution<                \
                cutlass::half_t, cutlass::layout::TensorNCHW, cutlass::half_t, \
                cutlass::layout::TensorNCHW, ElementOutput,                    \
                cutlass::layout::TensorNCHW, cutlass::half_t,                  \
                cutlass::layout::TensorNCHW, cutlass::half_t,                  \
                cutlass::conv::ConvType::kDepthwiseConvolution,                \
                cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,           \
                ThreadBlockShape, WarpShape, InstructionShape,                 \
                cutlass::epilogue::thread::BiasAddLinearCombination<           \
                        ElementOutput, 1, ElementAccumulator, ElementBias,     \
                        ElementCompute>,                                       \
                cutlass::conv::threadblock::                                   \
                        DepthwiseConvolutionFpropThreadblockSwizzle,           \
                stage, 1, 1, cutlass::conv::SpecialOptimizeDesc::NONE,         \
                cutlass::arch::OpMultiplyAdd,                                  \
                cutlass::conv::ImplicitGemmMode::GEMM_TN>;                     \
        EXPECT_TRUE(test::convolution::device::TestDepthwiseConvolution<       \
                    Convolution>());                                           \
    } while (0)

////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_Depthwise_Conv2dFprop_f16_f16_NCHW_tensor_op_f16,
     128x256x64_64x64x64) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<128, 256, 64>;
    using WarpShape = cutlass::gemm::GemmShape<64, 64, 64>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dFprop_f16_f16_NCHW_tensor_op_f16,
     256x128x64_64x64x64) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<256, 128, 64>;
    using WarpShape = cutlass::gemm::GemmShape<64, 64, 64>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dFprop_f16_f16_NCHW_tensor_op_f16,
     128x128x64_64x64x64) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<128, 128, 64>;
    using WarpShape = cutlass::gemm::GemmShape<64, 64, 64>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dFprop_f16_f16_NCHW_tensor_op_f16,
     64x128x64_32x64x64) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<64, 128, 64>;
    using WarpShape = cutlass::gemm::GemmShape<32, 64, 64>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dFprop_f16_f16_NCHW_tensor_op_f16,
     128x64x64_64x32x64) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<128, 64, 64>;
    using WarpShape = cutlass::gemm::GemmShape<64, 32, 64>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dFprop_f16_f16_NCHW_tensor_op_f16,
     64x64x64_32x32x64) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<64, 64, 64>;
    using WarpShape = cutlass::gemm::GemmShape<32, 32, 64>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dFprop_f16_f16_NCHW_tensor_op_f16,
     128x256x32_64x64x32) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<128, 256, 32>;
    using WarpShape = cutlass::gemm::GemmShape<64, 64, 32>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dFprop_f16_f16_NCHW_tensor_op_f16,
     256x128x32_64x64x32) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<256, 128, 32>;
    using WarpShape = cutlass::gemm::GemmShape<64, 64, 32>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dFprop_f16_f16_NCHW_tensor_op_f16,
     128x128x32_64x64x32) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<128, 128, 32>;
    using WarpShape = cutlass::gemm::GemmShape<64, 64, 32>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dFprop_f16_f16_NCHW_tensor_op_f16,
     256x64x32_64x64x32) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<256, 64, 32>;
    using WarpShape = cutlass::gemm::GemmShape<64, 64, 32>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dFprop_f16_f16_NCHW_tensor_op_f16,
     64x256x32_64x64x32) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<64, 256, 32>;
    using WarpShape = cutlass::gemm::GemmShape<64, 64, 32>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dFprop_f16_f16_NCHW_tensor_op_f16,
     64x128x32_32x64x32) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<64, 128, 32>;
    using WarpShape = cutlass::gemm::GemmShape<32, 64, 32>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dFprop_f16_f16_NCHW_tensor_op_f16,
     128x64x32_64x32x32) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<128, 64, 32>;
    using WarpShape = cutlass::gemm::GemmShape<64, 32, 32>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dFprop_f16_f16_NCHW_tensor_op_f16,
     64x64x32_32x32x32) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<64, 64, 32>;
    using WarpShape = cutlass::gemm::GemmShape<32, 32, 32>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

////////////////////////////////////////////////////////////////////////////////
#endif
