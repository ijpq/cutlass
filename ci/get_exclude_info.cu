
#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int count;
    hipGetDeviceCount(&count);
    if (count == 0) {
        throw std::runtime_error("No GPU Found");
    }
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::string output{};
    if (prop.major < 8) {
        output = "*SM80*";
    }
    if (prop.major < 7) {
        output += ":*SM70*";
    }
    std::cout << output;
    return 0;
}
